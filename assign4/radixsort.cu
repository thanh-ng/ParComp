
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <math.h> 



__global__ void copyByIndex(int *out, int *in, int *index, int n){

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < n){
        out[index[tid]] = in[tid]; 
    }
}


__global__ void copyByIdentity(int *out, int *in, int n){

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < n){
        out[tid] = in[tid]; 
    }
}

// Compute exclusive prefix sum 
__global__ void scan(int *d_out, int *d_in_array, int *SUMS, int *INCR, int *bias, int d_pow, int nBlocks, int n){
        // When using dynamically allocated shared memory, only one pointer to the allocated space will be given to the kernel code. If you want to divide up that space, you must do so yourself

	extern __shared__ int temp[]; // 4* blockSize , each block will allocate its own shared memory        

	int tid = blockIdx.x*blockDim.x + threadIdx.x; // thread id across blocks 
	//int bid = blockIdx.x; 
	// bool DEBUG = false; 
    int offset = 1; // offset for each iteration of Scan, located in the global memory 
	int blockSize = n / nBlocks; // number of elements to be processed in a block, blockSize / 2 = blockDim, REQ: power of 2
	int local_tid = (2*tid) % blockSize; 
	int bid = tid / (blockSize / 2); 

	//int bid = tid / (blockSize / 2); 	
	int t_index_zero, t_index_one;         
	int offset2; 

    // int my_monitor = 1; 
    // *bias = 0;


    temp[blockSize + local_tid ] =  (d_in_array[2*tid] / d_pow ) % 2 ;
    temp[blockSize + local_tid  + 1] = (d_in_array[2*tid + 1] / d_pow ) % 2;

    temp[local_tid] = 1 - (d_in_array[2*tid] / d_pow ) % 2 ;;
    temp[local_tid + 1] = 1 - (d_in_array[2*tid + 1] / d_pow ) % 2;

    temp[2*blockSize + local_tid] = (d_in_array[2*tid] / d_pow ) % 2; 
    temp[2*blockSize + local_tid +1] = (d_in_array[2*tid + 1] / d_pow ) % 2;
		
		//INCR[bid] = 0; 
		
		// MAYBE the copy here need to complete before proceeding to the next steps!
		//============= DEBUG
		//if(DEBUG){
		//    printf("INPUT:    [Block #%d] temp[%d] = %d, temp[%d] = %d\n", bid,  2*tid, temp[local_tid], 2*tid + 1, temp[local_tid+1]);
		//}
		//===================

        	// Sweep up
    for (int d = blockSize>>1; d > 0; d >>=1){
        __syncthreads();
            if (tid % ( blockSize / 2 ) < d) {
                int ai = offset * ( local_tid + 1) - 1;
                int bi = offset * ( local_tid + 2) -1;

                temp[bi] += temp[ai];

				temp[blockSize + bi] += temp[blockSize + ai];
       		}
            	offset *= 2;
    }

        	// for exclusive prefix sum 
    if (tid % (blockSize/2) == 0 ) { 
        // *bias += temp[blockSize-1]; 
        SUMS[bid] = temp[2*blockSize-1]; 
		//NZ[bid] = temp[blockSize-1]; 
			//temp[3*blockSize] = blockSize - temp[2*blockSize-1]; 
		temp[blockSize-1] = 0; 
		temp[2*blockSize-1] = 0;
	}

     __syncthreads(); 
         
         
         



    //======= Scan SUMS ==========
    // ASSUMPTION: nBlocks <= blockSize so that SUMS can be processed by a single block. 
    // In practice, blockSize = 1024. With this assumption, the max length of array input is 2**20; 

	offset2 = 1;
		//if ( 2*tid + 1 <= nBlocks -1){ 
	temp[3*blockSize + local_tid] = SUMS[local_tid]; 
	temp[3*blockSize + local_tid + 1] = SUMS[local_tid + 1]; 

	if(tid % (blockSize/2) == 0){ // zero padding
		for(int i = nBlocks; i < blockSize; i++){
			temp[3*blockSize + i] = 0;
		}
	}

    // printf("BEFORE\n");
    // printf("Block #%d, temp[3*blockSize + %d] = %d, temp[3*blockSize + %d] = %d \n", bid, local_tid, temp[3*blockSize + local_tid],local_tid + 1, temp[3*blockSize + local_tid + 1] );

    //__syncthreads(); 
        
    // Sweep up SUMS
	for(int d = blockSize>>1; d > 0; d >>= 1){
		__syncthreads(); 
			
    	if (tid % ( blockSize / 2 ) < d) {
        	int ai = offset2 * ( local_tid + 1) - 1;
        	int bi = offset2 * ( local_tid + 2) -1;

        	temp[3*blockSize + bi] += temp[3*blockSize + ai];
                        
   		}
    	offset2 *= 2;

	}

	if (tid % ( blockSize / 2 ) == 0 ) {  
		*bias = n - temp[3*blockSize + nBlocks-1];
		//printf("tid = %d, bias = %d\n", tid, *bias);
		temp[4*blockSize - 1] = 0;
	}

    // Sweep down SUMS
	for(int d = 1; d < blockSize; d *= 2){
    	offset2 >>= 1;
    	__syncthreads();

    	if (tid % (blockSize / 2)  < d) {
        	int ai = offset2 * (local_tid +1) - 1;
        	int bi = offset2 * (local_tid +2) - 1;

        	int t = temp[3*blockSize + ai];
        	temp[3*blockSize + ai] = temp[3*blockSize + bi];
        	temp[3*blockSize + bi] += t;
    	}
	}


	__syncthreads();

    // printf("AFTER\n");
    // printf("Block #%d, temp[3*blockSize + %d] = %d, temp[3*blockSize + %d] = %d \n", bid, local_tid, temp[3*blockSize + local_tid],local_tid + 1, temp[3*blockSize + local_tid + 1] );

    // if(tid % (blockSize/2) == 0){
    // //    printf("INCR[%d] = %d\n", bid,temp[3*blockSize + bid] ) ;
    //  //    INCR[bid] = temp[3*blockSize + bid];
    //     temp[4*blockSize] = temp[3*blockSize + bid]; 
    // }

    //if ( 2*tid + 1 <= nBlocks -1){

    //    INCR[2*tid] = temp[3*blockSize + 2*tid]; 
    //    INCR[2*tid + 1] = temp[3*blockSize + 2*tid + 1]; 
    //}

    //INCR[local_tid] = temp[3*blockSize + local_tid];
    //INCR[local_tid+1] = temp[3*blockSize + local_tid + 1];

    //MY_INCR[2*tid] = temp[3*blockSize + local_tid];
    //MY_INCR[2*tid+1] = temp[3*blockSize + local_tid + 1];
        	
	//__syncthreads();

	//}
	
	//if(2*tid + 1 <= nBlocks -1){
	 //   printf("INCR[%d] = %d, INCR[%d] = %d\n", 2*tid, INCR[2*tid], 2*tid +1, INCR[2*tid +1]);
	//}
	/*
    if(tid == 0){
	    printf("INCR: \n");
	    for(int i = 0; i < blockSize -1; i++){
	        printf("%d,", INCR[i]);
	        }
	        printf("%d\n", INCR[blockSize-1]);
	}
	*/
	        //==========================

        	// Sweep down 
    	for(int d = 1; d < blockSize; d *= 2){
            	offset >>= 1;
            	__syncthreads();

            	if (tid % (blockSize / 2)  < d) {
                    int ai = offset * (local_tid + 1) - 1;
                    int bi = offset * (local_tid + 2) - 1;

                    int t0 = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t0;
                    
                int t1 = temp[blockSize + ai];
                    temp[blockSize + ai] = temp[blockSize + bi];
                    temp[blockSize + bi] += t1;
           		 }
    	 	}


        // if(tid % (blockSize / 2) == 0){
        //     INCR[0] = 0; 
        //     *bias = SUMS[0]; 
        //     for(int i = 1; i < nBlocks; i++){
        //         INCR[i] = INCR[i-1] + SUMS[i-1]; 
        //         *bias += SUMS[i]; 
        //     }
            
        // }

    	__syncthreads();


        // if(tid == 0){
        //     printf("INCR: \n");
        //     for(int i = 0; i < nBlocks -1; i++){
        //         printf("%d,", INCR[i]);
        //         }
        //         printf("%d\n", INCR[nBlocks-1]);
                
        //     // printf("bias = %d\n", *bias);
        // }
        
    	
    	//==============DEBUG============================
    	// Check temp 
        // if(DEBUG){
    	// printf("OUTPUT:    [Block #%d] temp[%d] = %d, temp[%d] = %d\n", bid,  2*tid, temp[local_tid], 2*tid + 1, temp[local_tid+1]);
    	
    	// // Check INCR 
    	// if(tid == 0){
    	//     printf("INCR: \n");
    	//     for(int i = 0; i < nBlocks -1; i++){
    	//         printf("%d,", INCR[i]);
    	//         }
    	//         printf("%d\n", INCR[nBlocks-1]);
    	        
    	//     //printf("bias = %d\n", *bias);
    	// }
    	
    	// Check bias
    	
    	//===============================================
        // }
    	t_index_zero = (temp[local_tid] + bid*blockSize - temp[3*blockSize + bid])* (1 - temp[2*blockSize + local_tid]) + (temp[blockSize + local_tid] + temp[3*blockSize + bid] + *bias) * temp[2*blockSize + local_tid] ;
    	t_index_one = (temp[local_tid + 1] + bid*blockSize - temp[3*blockSize + bid])* (1 - temp[2*blockSize + local_tid + 1]) + (temp[blockSize + local_tid + 1] + temp[3*blockSize + bid] + *bias)* temp[2*blockSize + local_tid + 1];

        // t_index_one: =f(tid), thus each thread has a different t_index_one



        //my_monitor = tid; 
        //printf("bid = %d, tid = %d, my_monitor = %d\n", bid, tid, my_monitor);
        // d_out[t_index_zero] = d_in_array[2*tid];
        // d_out[t_index_one] = d_in_array[2*tid + 1]; 
    	
        // t_index_zero = *bias; //(temp[local_tid] + bid*blockSize - INCR[bid])* (1 - temp[2*blockSize + local_tid]) ;
        // t_index_one = *bias; //(temp[local_tid + 1] + bid*blockSize - INCR[bid])* (1 - temp[2*blockSize + local_tid + 1]);

        // printf("bid = %d, tid = %d, t_index_zero = %d, t_index_one = %d\n", bid, tid, t_index_one, t_index_zero);


        d_out[2*tid] = t_index_zero;
        d_out[2*tid + 1] = t_index_one; 
        //@(TODO): something wrong with indices for zero! More specifically, INCR[bid]



}

int main(int argc, char* argv[]){
    if(argc < 4){
        printf("Usage: %s seed N print_option\n", argv[0]);
        exit(1);
    }
    int SEED = atoi(argv[1]);
    int M = atoi(argv[2]);
    int PRINT = atoi(argv[3]); // print the output if 1, do not print if 0

    int N = pow(2, ceil( log2((float) M))); 
    int *in_array = (int*) malloc(N * sizeof(int));
    //printf("next = %d\n", *next);
    
    /* Initialize the random number generator for the given SEED */
    srand(SEED);
    /* Generate N pseudo-random integers in the interval [0, RAND_MAX] */
    for (int i = 0; i < M; i++)
        in_array[i] = rand();

    for(int i = M; i < N; i++)
        in_array[i] = 0; 

    // if(PRINT){
    //     printf("INPUT: \n");
    //     for (int i = 0; i < 20; i++)
    //         printf("%d\n", in_array[i]);

    // }

    // const int N = 16; // number of elements
	const int B = 1024; // number of elements to be processed in each block. REQ: power of 2 
	// N / B: number of blocks, with B/2 threads per block
    int nBlocks = N / B; 
        // Input

	//int in_array[N] = { 5, 6, 2, 7, 1, 0, 4, 3 };
	//int in_array[N] = {1804289383, 846930886, 1681692777, 1714636915, 1957747793, 424238335, 719885386, 1649760492};
	// int in_array[N] = {834, 86, 77, 15, 93, 35, 84, 92};
    // int in_array[N] = {5, 13, 6, 14, 2, 8, 12,  7, 15, 1, 0, 4, 9, 10, 3, 11};

    // int in_array[N] = {84, 92, 77, 93, 834, 86, 15, 35};

	int bit_width = 32; 
 
	int *d_out,*d_in_array;
 
    int index[N]; 
    int temp_in[N];

	int *SUMS; // To save the same of the array carried out by each block
	int *INCR; // Scanned result of SUMS

	int *bias; 
	int d_pow = 1; 



    int *d_index, *d_temp_in; 
    hipMalloc( (void **) &d_index, N*sizeof(int));
    hipMalloc( (void **) &d_temp_in, N*sizeof(int));


    hipMalloc( (void **) &d_out, N*sizeof(int));
    hipMalloc( (void **) &d_in_array, N*sizeof(int)); 
    hipMalloc( (void **) &SUMS, nBlocks*sizeof(int));
    hipMalloc( (void **) &INCR, nBlocks*sizeof(int));
    hipMalloc( (void **) &bias, sizeof(int));



    struct timeval start, end;
    gettimeofday(&start, 0);
    for (int i = 0; i < bit_width; i++){
    	hipMemcpy( d_in_array, in_array, N*sizeof(int), hipMemcpyHostToDevice);
    	scan<<< nBlocks , B / 2, (4*B + 1)*sizeof(int)>>>(d_out, d_in_array, SUMS, INCR, bias, d_pow, nBlocks, N); 
    	hipMemcpy( index, d_out, N*sizeof(int), hipMemcpyDeviceToHost);

        // cudaThreadSynchronize();

        hipMemcpy( d_in_array, in_array, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy( d_index, index, N*sizeof(int), hipMemcpyHostToDevice);
        copyByIndex<<<nBlocks , B >>>(d_temp_in, d_in_array, d_index, N); 
        hipMemcpy(temp_in, d_temp_in,  N*sizeof(int), hipMemcpyDeviceToHost);
        // for (int i = 0; i < N; i++){
        //     temp_in[index[i]] = in_array[i];
        // }

        hipMemcpy( d_temp_in, temp_in , N*sizeof(int), hipMemcpyHostToDevice);
        copyByIdentity<<<nBlocks , B >>>(d_in_array, d_temp_in, N); 
        hipMemcpy(in_array, d_in_array,  N*sizeof(int), hipMemcpyDeviceToHost);


        // for (int i = 0; i < N; i++){
        //     in_array[i] = temp_in[i];
        // }

        // for (int i = 0; i < N; i++){
        //     printf("%d ", in_array[i]);
        // }
        // printf("\n");

    	d_pow *= 2; 
    }

    gettimeofday(&end, 0);

       // Print s after sorted 
    // for (int i = 0; i < N; i++){
    //         printf("%d\n", in_array[i]);
    // }
	


    if(PRINT){
        printf("Output: \n");
        for (int i = 0; i < 20; i++)
            printf("%d\n", in_array[N - M + i]);

    }
    else{
        printf("Sorting Time: %lld sec.\n", end.tv_sec - start.tv_sec);
    }


	hipFree( d_in_array); 
	hipFree( d_out);
	hipFree( SUMS); 
	hipFree( INCR); 
	hipFree( bias); 
        return 0;
}
      
